/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <include/cufhe_gpu.cuh>
#include <include/bootstrap_gpu.cuh>

namespace cufhe {

void Initialize(const PubKey& pub_key) {
  BootstrappingKeyToNTT(pub_key.bk_);
  KeySwitchingKeyToDevice(pub_key.ksk_);
}

void CleanUp() {
  DeleteBootstrappingKeyNTT();
  DeleteKeySwitchingKey();
}

inline void CtxtCopyH2D(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_device_->data(),
                  c.lwe_sample_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyHostToDevice,
                  st.st());
}

inline void CtxtCopyD2H(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_->data(),
                  c.lwe_sample_device_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyDeviceToHost,
                  st.st());
}

template <int32_t mu, int32_t space>
constexpr Torus ModSwitchToTorusConst() {
  //static const uint64_t gap = ((0x1UL << 63) / space) * 2;
  return int32_t((uint64_t(mu) * ((0x1UL << 63) / space * 2)) >> 32);
}

class NandGate {
 public:
  static __device__ inline Torus a(Torus x, Torus y) { return 0-x-y; }
  static __device__ inline Torus b(Torus x, Torus y) { return kFix+a(x, y); }
 //private:
  static const Torus kFix = ModSwitchToTorusConst<1, 8>();
};

void Nand(Ctxt& out,
          const Ctxt& in0,
          const Ctxt& in1,
          Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  if (fix != ((NandGate*)nand_gate)->kFix)
    std::cout<< "wrong\t" << fix << "\t" << ((NandGate*)nand_gate)->kFix <<std::endl;
/*  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 - in0.lwe_sample_->data()[i]
                                   - in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
*/
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  //Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
  //CtxtCopyH2D(out, st);
  //Bootstrap(out.lwe_sample_device_, out.lwe_sample_device_, mu, st.st());
  //NandBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_, in1.lwe_sample_device_, mu, fix, st.st());
  Bootstrap<NandGate>(out.lwe_sample_device_, in0.lwe_sample_device_, in1.lwe_sample_device_, mu, st.st());
  CtxtCopyD2H(out, st);
}
/*
void Or(Ctxt& out,
        const Ctxt& in0,
        const Ctxt& in1,
        Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 + in0.lwe_sample_->data()[i]
                                   + in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void And(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 + in0.lwe_sample_->data()[i]
                                   + in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Nor(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 - in0.lwe_sample_->data()[i]
                                   - in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Xor(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 4);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 + 2 * in0.lwe_sample_->data()[i]
                                   + 2 * in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Xnor(Ctxt& out,
          const Ctxt& in0,
          const Ctxt& in1,
          Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 4);
  for (int i = 0; i <= in0.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = 0 - 2 * in0.lwe_sample_->data()[i]
                                   - 2 * in1.lwe_sample_->data()[i];
  out.lwe_sample_->b() += fix;
  Bootstrap(out.lwe_sample_, out.lwe_sample_, mu, st.st());
}

void Not(Ctxt& out,
         const Ctxt& in,
         Stream st) {
  for (int i = 0; i <= in.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = -in.lwe_sample_->data()[i];
}

void Copy(Ctxt& out,
          const Ctxt& in,
          Stream st) {
  for (int i = 0; i <= in.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = in.lwe_sample_->data()[i];
}
*/
} // namespace cufhe
